#include "hip/hip_runtime.h"
#include "chest_sim.cuh"
#include "carver_reversal.cuh"

#include <chrono>
#include <cstdio>
#include <vector>
#include <cstdlib>

#include <thread>
#include <mutex>

// --------------------------------------------------------------------
// global program params

constexpr uint64_t CARVER_SEED = 137099342588438ULL;
constexpr int MIN_CHESTS = 4;

constexpr int BATCH_SIZE = 100;
constexpr int CHUNKS_ON_AXIS = 60'000'000 / 16;
constexpr int TASKS_ON_AXIS = CHUNKS_ON_AXIS / BATCH_SIZE;
constexpr uint64_t MAX_TASK_ID = (uint64_t)TASKS_ON_AXIS * TASKS_ON_AXIS;

// --------------------------------------------------------------------
// multithreaded carver reversal

typedef struct Result Result;
struct Result {
    uint64_t worldseed;
    int chunk_x;
    int chunk_z;
};

std::vector<Result> carver_step_results;
std::mutex result_mutex;

void reverse_carver(int x, int z, ReversalOutput& out) {
    reverseCarverSeedCPU(CARVER_SEED, x, z, &out);
}

void carver_reversal_worker(int x_min, int x_max, int z) {
    ReversalOutput out = { 0 };

    for (int x = x_min; x < x_max; ++x) {
        out.resultCount = 0;
        reverse_carver(x, z, out);
        {
            std::lock_guard<std::mutex> lock(result_mutex);
            for (int i = 0; i < out.resultCount; i++)
                carver_step_results.push_back({out.results[i], x, z});
        } 
    }
}

// --------------------------------------------------------------------
// gpu worldseed bruteforce

constexpr int MAX_WORLDSEED_RESULTS = 64;
__managed__ int worldseedResultCount = 0;
__managed__ Result worldseedResults[MAX_WORLDSEED_RESULTS];

extern __device__ int countChests(Xoroshiro*);
__global__ void bruteforceWorldseeds(const uint64_t structure_seed, const int x, const int z) {
    uint64_t upper16 = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t worldseed = structure_seed | (upper16 << 48);

    Xoroshiro xr;
    xSetDecoratorSeed(&xr, worldseed, x<<4, z<<4, 30001);
    int chests = countChests(&xr);
    if (chests >= MIN_CHESTS) {
        int ix = atomicAdd(&worldseedResultCount, 1);
        if (ix < MAX_WORLDSEED_RESULTS)
            worldseedResults[ix] = {worldseed, x, z};
    }
}

__host__ void launchBruteforce() {
    worldseedResultCount = 0;
    for (auto& result : carver_step_results) {
        bruteforceWorldseeds <<< 256, 256 >>> (result.worldseed, result.chunk_x, result.chunk_z);
    }
    carver_step_results.clear();
    CUDA_CHECK(hipGetLastError());
}

__host__ void processResults() {
    for (int i = 0; i < worldseedResultCount; i++) {
        printf("%lld %d %d\n", worldseedResults[i].worldseed, worldseedResults[i].chunk_x, worldseedResults[i].chunk_z);
    }
}

// --------------------------------------------------------------------

int parse_args(int argc, char* argv[], int& task_min, int& task_max, int& thread_count) {
    task_min = task_max = thread_count = -1;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--start") == 0 && i != argc-1)
            task_min = atoi(argv[i+1]);
        else if (strcmp(argv[i], "--end") == 0 && i != argc-1)
            task_max = atoi(argv[i+1]);
        else if (strcmp(argv[i], "--threads") == 0 && i != argc-1)
            thread_count = atoi(argv[i+1]);
    }

    if (task_min == -1 || task_max == -1 || thread_count == -1) {
        fprintf(stderr, "Usage: (executable) --start [TASK_ID] --end [TASK_ID] --threads [NUM_THREADS]\n");
        return 1;
    }

    return 0;
}

int main(int argc, char* argv[]) {
    int task_min, task_max, thread_count;
    if (parse_args(argc, argv, task_min, task_max, thread_count)) {
        fprintf(stderr, "Launch failed.\n");
        return 1;
    }

    int current_task = task_min;
    int current_task_z = 0;
    int subtasks_total = (task_max - task_min) * BATCH_SIZE;

    while (current_task < task_max) {
        std::vector<std::thread> threads;

        int subtasks_done = (current_task - task_min) * BATCH_SIZE + current_task_z;
        fprintf(stderr, "--- progress: %d / %d subtasks done\n", subtasks_done, subtasks_total);

        for (int i = 0; i < thread_count; i++) {
            const int tx = (current_task / TASKS_ON_AXIS) * BATCH_SIZE;
            const int tz = (current_task % TASKS_ON_AXIS) * BATCH_SIZE;
            threads.emplace_back(carver_reversal_worker, tx, tx + BATCH_SIZE, tz + current_task_z);
            current_task_z++;

            if (current_task_z >= BATCH_SIZE) {
                current_task_z = 0;
                current_task++;
                if (current_task >= task_max)
                    break;
            }
        }
        for (auto& t : threads) {
            t.join();
        }

        auto t0 = std::chrono::steady_clock::now();
        CUDA_CHECK(hipDeviceSynchronize());
        auto t1 = std::chrono::steady_clock::now();
        double msWaited = (t1 - t0).count() * 1e-6;
        fprintf(stderr, "--- waited %f ms for GPU\n", msWaited);

        processResults();
        launchBruteforce();
    }

    CUDA_CHECK(hipDeviceSynchronize());
    processResults();
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
