#include "hip/hip_runtime.h"
/*
This version of the checker runs the carver reversal on multiple CPU threads,
and delegates worldseed checks to the GPU. Because carver reversal will usually be 
the main bottleneck, this won't use the full potential of the GPU, but could be
enough for machines with powerful CPUs and weak GPUs.
*/

#include "chest_sim.cuh"
#include "carver_reversal.cuh"

#include <chrono>
#include <cstdio>
#include <vector>
#include <cstdlib>

#include <thread>
#include <mutex>

// --------------------------------------------------------------------
// global program params

constexpr uint64_t CARVER_SEED = 190383783165418ULL; //137099342588438ULL;
constexpr int MIN_CHESTS = 3;

constexpr int BATCH_SIZE = 100;
constexpr int CHUNKS_ON_AXIS = 60'000'000 / 16;
constexpr int TASKS_ON_AXIS = CHUNKS_ON_AXIS / BATCH_SIZE;
constexpr int TASK_COORD_OFFSET = TASKS_ON_AXIS / 2;
//constexpr uint64_t MAX_TASK_ID = (uint64_t)TASKS_ON_AXIS * TASKS_ON_AXIS;

// --------------------------------------------------------------------
// multithreaded carver reversal

typedef struct Result Result;
struct Result {
    uint64_t worldseed;
    int chunk_x;
    int chunk_z;
};

std::vector<Result> carver_step_results;
std::mutex result_mutex;

static bool trial_chamber_can_generate(int x, int z) {
    int x_in_region = (x%34 + 34) % 34;
    int z_in_region = (z%34 + 34) % 34;
    return x_in_region < 22 && z_in_region < 22;
}

static void reverse_carver(int x, int z, ReversalOutput& out) {
    reverseCarverSeed(CARVER_SEED, x, z, &out);
}

static void check_carver_result(Result res) {
    // check position
    int rx = (int)std::floor(res.chunk_x / 34.0);
    int rz = (int)std::floor(res.chunk_z / 34.0);

    uint64_t rand = 0;
    setRegionSeed(&rand, res.worldseed, rx, rz, 94251327);
    int cx = rx * 34 + nextInt(&rand, 22);
    int cz = rz * 34 + nextInt(&rand, 22);

    if (cx == res.chunk_x && cz == res.chunk_z)
    {
        std::lock_guard<std::mutex> lock(result_mutex);
        carver_step_results.push_back(res);
    }
}

static void carver_reversal_worker(int x_min, int x_max, int z) {
    ReversalOutput out = { 0 };

    for (int x = x_min; x < x_max; ++x) {
        out.resultCount = 0;
        if (!trial_chamber_can_generate(x, z))
            continue;

        reverse_carver(x, z, out);
        for (int i = 0; i < out.resultCount; i++)
            check_carver_result({out.results[i], x, z-1});
    }
}

// --------------------------------------------------------------------
// gpu worldseed bruteforce

constexpr int MAX_WORLDSEED_RESULTS = 64;
__managed__ int worldseedResultCount = 0;
__managed__ Result worldseedResults[MAX_WORLDSEED_RESULTS];

__global__ void bruteforceWorldseeds(const uint64_t structure_seed, const int x, const int z) {
    uint64_t upper16 = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t worldseed = structure_seed | (upper16 << 48);

    Xoroshiro xr;
    xSetDecoratorSeed(&xr, worldseed, x<<4, z<<4, 30001);
    int chests = countChests(&xr);
    if (chests >= MIN_CHESTS) {
        int ix = atomicAdd(&worldseedResultCount, 1);
        if (ix < MAX_WORLDSEED_RESULTS)
            worldseedResults[ix] = {worldseed, x, z};
    }
}

__host__ void launchBruteforce() {
    for (auto& result : carver_step_results) {
        bruteforceWorldseeds <<< 256, 256 >>> (result.worldseed, result.chunk_x, result.chunk_z);
    }
    carver_step_results.clear();
    CUDA_CHECK(hipGetLastError());
}

__host__ void processResults() {
    for (int i = 0; i < worldseedResultCount; i++) {
        printf("%lld  /tp %d 0 %d\n", worldseedResults[i].worldseed, worldseedResults[i].chunk_x*16, worldseedResults[i].chunk_z*16 + 128);
    }
}

// --------------------------------------------------------------------

int parse_args(int argc, char* argv[], int& task_min, int& task_max, int& thread_count) {
    task_min = task_max = thread_count = -1;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--start") == 0 && i != argc-1)
            task_min = atoi(argv[i+1]);
        else if (strcmp(argv[i], "--end") == 0 && i != argc-1)
            task_max = atoi(argv[i+1]);
        else if (strcmp(argv[i], "--threads") == 0 && i != argc-1)
            thread_count = atoi(argv[i+1]);
    }

    if (task_min == -1 || task_max == -1 || thread_count == -1) {
        fprintf(stderr, "Usage: (executable) --start [TASK_ID] --end [TASK_ID] --threads [NUM_THREADS]\n");
        return 1;
    }

    return 0;
}

int main(int argc, char* argv[]) {
    int task_min, task_max, thread_count;
    if (parse_args(argc, argv, task_min, task_max, thread_count)) {
        fprintf(stderr, "Launch failed.\n");
        return 1;
    }

    int current_task = task_min;
    int current_task_z = 0;
    int subtasks_total = (task_max - task_min) * BATCH_SIZE;
    int ix = 0;

    while (current_task < task_max) {
        std::vector<std::thread> threads;

        int subtasks_done = (current_task - task_min) * BATCH_SIZE + current_task_z;
        fprintf(stderr, "--- progress: %d / %d subtasks done\n", subtasks_done, subtasks_total);

        for (int i = 0; i < thread_count; i++) {
            const int tx = (current_task / TASKS_ON_AXIS) * BATCH_SIZE - TASK_COORD_OFFSET * BATCH_SIZE;
            const int tz = (current_task % TASKS_ON_AXIS) * BATCH_SIZE - TASK_COORD_OFFSET * BATCH_SIZE;
            if (trial_chamber_can_generate(0, current_task_z))
                threads.emplace_back(carver_reversal_worker, tx, tx + BATCH_SIZE, tz + current_task_z);
            else i--; // need to run the loop an additional time

            current_task_z++;
            if (current_task_z >= BATCH_SIZE) {
                current_task_z = 0;
                current_task++;
                if (current_task >= task_max)
                    break;
            }
        }
        for (auto& t : threads) {
            t.join();
        }

        if ((ix++ & 3) == 0) {
            CUDA_CHECK(hipDeviceSynchronize());
            processResults();
            worldseedResultCount = 0;
        }

        launchBruteforce();
    }

    CUDA_CHECK(hipDeviceSynchronize());
    processResults();
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
